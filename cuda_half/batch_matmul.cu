#include <iostream>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16 // Define block size for CUDA

// Kernel function to perform matrix multiplication for each batch using FP16
__global__ void batchMatMulFP16(half* A, half* B, half* C, int N, int batch_size) {
    int batch_idx = blockIdx.z; // Batch index
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row of the matrix
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column of the matrix
    
    if (row < N && col < N) {
        half value = __float2half(0.0f); // Initialize with 0 (in FP16)
        for (int k = 0; k < N; ++k) {
            value = __hadd(value, __hmul(A[batch_idx * N * N + row * N + k], B[batch_idx * N * N + k * N + col]));
        }
        C[batch_idx * N * N + row * N + col] = value;
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void loop(int N, int batch_size) {
    

    // Host memory allocations for FP16
    half *h_A, *h_B, *h_C;
    h_A = (half*)malloc(batch_size * N * N * sizeof(half));
    h_B = (half*)malloc(batch_size * N * N * sizeof(half));
    h_C = (half*)malloc(batch_size * N * N * sizeof(half));

    // Initialize host matrices with some values (using float-to-half conversion)
    for (int b = 0; b < batch_size; ++b) {
        for (int i = 0; i < N * N; ++i) {
            h_A[b * N * N + i] = __float2half(static_cast<float>(rand() % 10));
            h_B[b * N * N + i] = __float2half(static_cast<float>(rand() % 10));
        }
    }

    // Device memory allocations for FP16
    half *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc((void**)&d_A, batch_size * N * N * sizeof(half)), "Failed to allocate device memory for A");
    checkCudaError(hipMalloc((void**)&d_B, batch_size * N * N * sizeof(half)), "Failed to allocate device memory for B");
    checkCudaError(hipMalloc((void**)&d_C, batch_size * N * N * sizeof(half)), "Failed to allocate device memory for C");

    // Create CUDA events to measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start, 0);

    // Copy data from host to device
    checkCudaError(hipMemcpy(d_A, h_A, batch_size * N * N * sizeof(half), hipMemcpyHostToDevice), "Failed to copy A to device");
    checkCudaError(hipMemcpy(d_B, h_B, batch_size * N * N * sizeof(half), hipMemcpyHostToDevice), "Failed to copy B to device");

    // Record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate and print execution time
    float t_transfer = 0;
    hipEventElapsedTime(&t_transfer, start, stop);


    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE, batch_size);

    // Create CUDA events to measure time
    //cudaEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start, 0);

    // Launch kernel
    batchMatMulFP16<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N, batch_size);
    checkCudaError(hipGetLastError(), "Kernel launch failed");

     // Record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate and print execution time
    float t_execution = 0;
    hipEventElapsedTime(&t_execution, start, stop);


    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // Copy result back to host
    checkCudaError(hipMemcpy(h_C, d_C, batch_size * N * N * sizeof(half), hipMemcpyDeviceToHost), "Failed to copy C to host");

    // Print result
    /*
    for (int b = 0; b < batch_size; ++b) {
        std::cout << "Result for batch " << b << ":\n";
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << __half2float(h_C[b * N * N + i * N + j]) << " "; // Convert half to float for printing
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    */

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    std::cout << batch_size <<", " << N << ", " << t_transfer << ", " << t_execution << std::endl;

    return;
}

int main()
{
    int batch_sizes[6] = {1, 2, 4, 8, 16, 32};
    int shapes[4] = {1000, 2000, 3000, 4000};
    std::cout << "bs, shape, transfer, compute" << std::endl;
    for(auto batch_size: batch_sizes)
    {
        for(auto shape: shapes)
        {
            loop(shape, batch_size);
        }
    }
    return 0;
}