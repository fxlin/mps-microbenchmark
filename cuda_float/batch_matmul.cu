#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16 // Define block size for CUDA

// Kernel function to perform matrix multiplication for each batch
__global__ void batchMatMul(float* A, float* B, float* C, int N, int batch_size) {
    int batch_idx = blockIdx.z; // Batch index
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row of the matrix
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column of the matrix
    
    if (row < N && col < N) {
        float value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[batch_idx * N * N + row * N + k] * B[batch_idx * N * N + k * N + col];
        }
        C[batch_idx * N * N + row * N + col] = value;
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void loop(int N, int batch_size) {
    
    // Host memory allocations
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(batch_size * N * N * sizeof(float));
    h_B = (float*)malloc(batch_size * N * N * sizeof(float));
    h_C = (float*)malloc(batch_size * N * N * sizeof(float));

    // Initialize host matrices with some values
    for (int b = 0; b < batch_size; ++b) {
        for (int i = 0; i < N * N; ++i) {
            h_A[b * N * N + i] = static_cast<float>(rand() % 10);
            h_B[b * N * N + i] = static_cast<float>(rand() % 10);
        }
    }

    // Device memory allocations
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc((void**)&d_A, batch_size * N * N * sizeof(float)), "Failed to allocate device memory for A");
    checkCudaError(hipMalloc((void**)&d_B, batch_size * N * N * sizeof(float)), "Failed to allocate device memory for B");
    checkCudaError(hipMalloc((void**)&d_C, batch_size * N * N * sizeof(float)), "Failed to allocate device memory for C");


    // Create CUDA events to measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start, 0);

    // Copy data from host to device
    checkCudaError(hipMemcpy(d_A, h_A, batch_size * N * N * sizeof(float), hipMemcpyHostToDevice), "Failed to copy A to device");
    checkCudaError(hipMemcpy(d_B, h_B, batch_size * N * N * sizeof(float), hipMemcpyHostToDevice), "Failed to copy B to device");

    // Record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate and print execution time
    float t_transfer = 0;
    hipEventElapsedTime(&t_transfer, start, stop);
   
    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE, batch_size);

    // Create CUDA events to measure time
    //cudaEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start, 0);

    // Launch kernel
    batchMatMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N, batch_size);
    checkCudaError(hipGetLastError(), "Kernel launch failed");

    // Record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate and print execution time
    float t_execution = 0;
    hipEventElapsedTime(&t_execution, start, stop);
    

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy result back to host
    checkCudaError(hipMemcpy(h_C, d_C, batch_size * N * N * sizeof(float), hipMemcpyDeviceToHost), "Failed to copy C to host");

    // Print result
    /*
    for (int b = 0; b < batch_size; ++b) {
        std::cout << "Result for batch " << b << ":\n";
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << h_C[b * N * N + i * N + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    */

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    std::cout << batch_size <<", " << N << ", " << t_transfer << ", " << t_execution << std::endl;
    

    return;
}

int main()
{
    int batch_sizes[6] = {1, 2, 4, 8, 16, 32};
    int shapes[4] = {1000, 2000, 3000, 4000};
    std::cout << "bs, shape, transfer, compute" << std::endl;
    for(auto batch_size: batch_sizes)
    {
        for(auto shape: shapes)
        {
            loop(shape, batch_size);
        }
    }
    return 0;
}